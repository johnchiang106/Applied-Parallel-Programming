#include "hip/hip_runtime.h"
// Histogram Equalization

#include <wb.h>

#define HISTOGRAM_LENGTH 256
//@@ insert code here
// convert to unsigned char
__global__ void toUnsignedChar(float *input, unsigned char *output, int size) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < size) {
    output[id] = (unsigned char) (255*input[id]); 
  }
}
// cast back to float
__global__ void toFloat(unsigned char *input, float *output, int size) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < size) {
    output[id] = (float) (input[id]/255.0);
  }
}
// convert to gray
__global__ void toGray(unsigned char *input, unsigned char *output, int size) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  if (x < size) {
    output[x] = (unsigned char) (0.21*input[3*x] + 0.71*input[3*x+1] + 0.07*input[3*x+2]);
  }
}
// compute histogram
__global__ void histogram(unsigned char *input, unsigned int *output, int len) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  __shared__ unsigned int histo_s[HISTOGRAM_LENGTH];

  if (threadIdx.x < HISTOGRAM_LENGTH) {
    histo_s[threadIdx.x] = 0;
  }
  __syncthreads();

  if (idx < len) {
    int pos = input[idx];
    atomicAdd(&(histo_s[pos]), 1);
  }
  __syncthreads();

  if (threadIdx.x < HISTOGRAM_LENGTH) {
    atomicAdd(&(output[threadIdx.x]), histo_s[threadIdx.x]);
  }
}

// scan
__global__ void scan(unsigned int *histogram, float *cdf, int size) {
  __shared__ float XY[HISTOGRAM_LENGTH];

  int i = threadIdx.x;
  if (i < HISTOGRAM_LENGTH) XY[i] = histogram[i];
  if (i + blockDim.x < HISTOGRAM_LENGTH) XY[i+blockDim.x] = histogram[i+blockDim.x];

  for (int stride = 1; stride <= blockDim.x; stride *= 2) {
    __syncthreads();
    int index = (i+1) * 2 * stride - 1;
    if (index < HISTOGRAM_LENGTH) {
      XY[index] += XY[index - stride];
    }
  }

  for (int stride = ceil(HISTOGRAM_LENGTH/4.0); stride > 0; stride /= 2) {
    __syncthreads();
    int index = (i+1)*stride*2 - 1;
    if(index + stride < HISTOGRAM_LENGTH) {
      XY[index + stride] += XY[index];
    }
  }
  __syncthreads();
  if (i < HISTOGRAM_LENGTH) cdf[i] = ((float) (XY[i]*1.0)/size);
  if (i + blockDim.x < HISTOGRAM_LENGTH) cdf[i+blockDim.x] = ((float) (XY[i+blockDim.x]*1.0)/size);
}

// histogram equalization function
__global__ void equalize(unsigned char *inout, float *cdf, int size) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < size) {
    float equalized = 255.0*(cdf[inout[id]]-cdf[0])/(1.0-cdf[0]);
    inout[id] = (unsigned char) (min(max(equalized, 0.0), 255.0));
  }
}

int main(int argc, char **argv) {
  wbArg_t args;
  int imageWidth;
  int imageHeight;
  int imageChannels;
  wbImage_t inputImage;
  wbImage_t outputImage;
  float *hostInputImageData;
  float *hostOutputImageData;
  const char *inputImageFile;

  //@@ Insert more code here
  float   *deviceImageFloat;
  unsigned char *deviceImageChar;
  unsigned char *deviceImageCharGrayScale;
  unsigned int  *deviceImageHistogram;
  float   *deviceImageCDF;

  args = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 0);

  wbTime_start(Generic, "Importing data and creating memory on host");
  inputImage = wbImport(inputImageFile);
  hostInputImageData = wbImage_getData(inputImage);
  imageWidth = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);
  imageChannels = wbImage_getChannels(inputImage);
  outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);
  hostOutputImageData = wbImage_getData(outputImage);
  wbTime_stop(Generic, "Importing data and creating memory on host");

  // Allocating GPU memory
  hipMalloc((void **)&deviceImageFloat, imageWidth*imageHeight*imageChannels*sizeof(float));
  hipMalloc((void **)&deviceImageChar, imageWidth*imageHeight*imageChannels*sizeof(unsigned char));
  hipMalloc((void **)&deviceImageCharGrayScale, imageWidth*imageHeight*sizeof(unsigned char));
  hipMalloc((void **)&deviceImageHistogram, HISTOGRAM_LENGTH*sizeof(unsigned int));
  //hipMemset((void *) deviceImageHistogram, 0, HISTOGRAM_LENGTH *sizeof(unsigned int));
  hipMalloc((void **)&deviceImageCDF, HISTOGRAM_LENGTH*sizeof(float));
  
  // Copy data to GPU
  hipMemcpy(deviceImageFloat, hostInputImageData, 
             imageWidth*imageHeight*imageChannels*sizeof(float),hipMemcpyHostToDevice);
  

  // convert to unsigned char
  dim3 dimGrid1(ceil(imageWidth*imageHeight*imageChannels/512.0), 1, 1);
  dim3 dimBlock1(512,1,1);
  toUnsignedChar<<<dimGrid1,dimBlock1>>>(deviceImageFloat, 
                                         deviceImageChar, imageWidth*imageHeight*imageChannels);
  hipDeviceSynchronize();
  // convert to gray
  dim3 dimGrid2(ceil(imageWidth*imageHeight/512.0), 1, 1);
  dim3 dimBlock2(512,1,1);
  toGray<<<dimGrid2,dimBlock2>>>(deviceImageChar, 
                                 deviceImageCharGrayScale, imageWidth*imageHeight);
  hipDeviceSynchronize();
  // compute histogram
  dim3 dimGrid3(ceil(imageWidth*imageHeight/256.0), 1, 1);
  dim3 dimBlock3(256,1,1);
  histogram<<<dimGrid3,dimBlock3>>>(deviceImageCharGrayScale, 
                                 deviceImageHistogram, imageWidth*imageHeight);
  hipDeviceSynchronize();
  // scan
  dim3 dimGrid4(1, 1, 1);
  dim3 dimBloc4(128,1,1);
  scan<<<dimGrid4, dimBloc4>>>(deviceImageHistogram, deviceImageCDF, imageWidth*imageHeight);
  hipDeviceSynchronize();
  // histogram equalization function
  dim3 dimGrid5(ceil(imageWidth*imageHeight*imageChannels/512.0), 1, 1);
  dim3 dimBlock5(512,1,1);
  equalize<<<dimGrid5,dimBlock5>>>(deviceImageChar, 
                                  deviceImageCDF, imageWidth*imageHeight*imageChannels);
  hipDeviceSynchronize();
  // cast to float
  dim3 dimGrid6(ceil(imageWidth*imageHeight*imageChannels/512.0), 1, 1);
  dim3 dimBlock6(512,1,1);
  toFloat<<<dimGrid6,dimBlock6>>>(deviceImageChar, 
                                 deviceImageFloat, imageWidth*imageHeight*imageChannels);
  hipDeviceSynchronize();
  hipMemcpy(hostOutputImageData, deviceImageFloat,
             imageWidth * imageHeight * imageChannels * sizeof(float), hipMemcpyDeviceToHost);

  // Check Solution 
  wbImage_setData(outputImage, hostOutputImageData);
  wbSolution(args, outputImage);

  // Free GPU Memory 
  hipFree(deviceImageFloat);
  hipFree(deviceImageChar);
  hipFree(deviceImageCharGrayScale);
  hipFree(deviceImageHistogram);
  hipFree(deviceImageCDF);
  // Free CPU Memory
  free(hostInputImageData);
  free(hostOutputImageData);
  return 0;
}
