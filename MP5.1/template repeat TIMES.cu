#include "hip/hip_runtime.h"
// MP 5.1 Reduction
// Given a list of length n
// Output its sum = lst[0] + lst[1] + ... + lst[n-1];

#include <wb.h>

#define BLOCK_SIZE 512 //@@ You can change this
#define TIMES 2

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)
  
__global__ void total(float *input, float *output, int len) {
  //@@ Load a segment of the input vector into shared memory
  //@@ Traverse the reduction tree
  //@@ Write the computed sum of the block to the output vector at the
  //@@ correct index
  __shared__ float partialSum[BLOCK_SIZE * TIMES];
  int dx = blockDim.x, bx = blockIdx.x, tx = threadIdx.x;
  unsigned int start = TIMES * bx * dx;
  for(int i = 0; i < TIMES; ++i){
    if(start + tx + dx*i < len)
      partialSum[tx + dx*i] = input[start + tx + dx*i];
    else
      partialSum[tx + dx*i] = 0;
  }
  //@@ need other implementation if TIMES != 2
  for (unsigned int stride = dx; stride >= 1; stride /= 2) {
    __syncthreads();
    if (tx < stride)
      partialSum[tx] += partialSum[tx + stride];
  }
  // output[bx*dx+tx] = partialSum[tx];
  output[bx] = partialSum[0];
}

int main(int argc, char **argv) {
  int ii;
  wbArg_t args;
  float *hostInput;  // The input 1D list
  float *hostOutput; // The output list
  float *deviceInput;
  float *deviceOutput;
  int numInputElements;  // number of elements in the input list
  int numOutputElements; // number of elements in the output list

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput =
      (float *)wbImport(wbArg_getInputFile(args, 0), &numInputElements);

  // numOutputElements = numInputElements / (BLOCK_SIZE * 2);
  // if (numInputElements % (BLOCK_SIZE * 2)) numOutputElements++;
  // numOutputElements = (numInputElements - 1) / (BLOCK_SIZE << 1) + 1;
  numOutputElements = (numInputElements - 1) / (BLOCK_SIZE * TIMES) + 1;
  // numOutputElements *= BLOCK_SIZE;
  hostOutput = (float *)malloc(numOutputElements * sizeof(float));

  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The number of input elements in the input is ",
        numInputElements);
  wbLog(TRACE, "The number of output elements in the input is ",
        numOutputElements);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  hipMalloc((void **) &deviceInput, numInputElements * sizeof(float));
  hipMalloc((void **) &deviceOutput, numOutputElements * sizeof(float));

  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceInput, hostInput, numInputElements * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(deviceOutput, hostOutput, numOutputElements * sizeof(float), hipMemcpyHostToDevice);

  wbTime_stop(GPU, "Copying input memory to the GPU.");
  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid(numInputElements/BLOCK_SIZE,1,1);
  if (numInputElements%BLOCK_SIZE) DimGrid.x++;
  dim3 DimBlock(BLOCK_SIZE,1,1);


  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  total<<<DimGrid,DimBlock>>>(deviceInput, deviceOutput, numInputElements);

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostOutput, deviceOutput, numOutputElements * sizeof(float), hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying output memory to the CPU");

  /***********************************************************************
   * Reduce output vector on the host
   * NOTE: One could also perform the reduction of the output vector
   * recursively and support any size input.
   * For simplicity, we do not require that for this lab!
   ***********************************************************************/
  for (ii = 1; ii < numOutputElements; ii++) {
    hostOutput[0] += hostOutput[ii];
  }

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceInput); hipFree (deviceOutput);

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, 1);

  free(hostInput);
  free(hostOutput);

  return 0;
}
