#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_WIDTH 16
// __constant__ float MASK[MASK_WIDTH*MASK_WIDTH];
__global__ void conv_forward_kernel(float *output, const float *input, const float *mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    Batch - batch_size (number of images in x)
    Map_out - number of output feature maps
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */

    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    int W_size = Width_out/TILE_WIDTH;
    // int H_size = Height_out/TILE_WIDTH;
    if (Width_out%TILE_WIDTH)  W_size++;
    // if (Height_out%TILE_WIDTH)  H_size++;
    // int W_size = ceil((Width_out * 1.0) / TILE_WIDTH);
    // (void)Height_out; // silence declared but never referenced warning. remove this line when you start working
    // (void)Width_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

    #define in_4d(b, c, h, w) input[(b) * (Channel * Height * Width) + (c) * (Height * Width) + (h) * (Width) + w]
    #define mask_4d(m, c, h, w) mask[(m) * (Channel * K * K) + (c) * (K * K) + (h) * (K) + w]
    #define out_4d(b, m, h, w) output[(b) * (Map_out * Height_out * Width_out) + (m) * (Height_out * Width_out) + (h) * (Width_out) + w]

    // Insert your GPU convolution kernel code here
    int m = blockIdx.x;
    int h = (blockIdx.y / W_size) * TILE_WIDTH + threadIdx.y;
    int w = (blockIdx.y % W_size) * TILE_WIDTH + threadIdx.x;
    int b = blockIdx.z;
    if(h < Height_out && w < Width_out){
        float acc = 0.0f;
        for (int c = 0; c < Channel; c++) { // sum over all input channels
            for (int p = 0; p < K; p++) // loop over KxK filter
                for (int q = 0; q < K; q++)
                    // if(h+p < height, w+q < width)
                        acc += in_4d(b, c, h+p, w+q) * mask_4d(m, c, p, q);
        }
        out_4d(b, m, h, w) = acc;
    }

    #undef out_4d
    #undef in_4d
    #undef mask_4d
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    int W_out = Width - K + 1, H_out = Height - K + 1;
    // Allocate memory and copy over the relevant data structures to the GPU
    // hipMemcpyToSymbol(HIP_SYMBOL(MASK), host_mask, Map_out * K * K * sizeof(float));
    hipMalloc((void **) device_input_ptr, Batch * Channel * Height * Width * sizeof(float));
    hipMalloc((void **) device_mask_ptr, Channel * Map_out * K * K * sizeof(float));
    hipMalloc((void **) device_output_ptr, Batch * Map_out * W_out * H_out * sizeof(float));
    
    // We pass double pointers for you to initialize the relevant device pointers, which are passed to the other two functions.
    hipMemcpy(*device_input_ptr, host_input, Batch * Channel * Height * Width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, Channel * Map_out * K * K * sizeof(float), hipMemcpyHostToDevice);

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }

}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    int W_out = Width - K + 1, H_out = Height - K + 1;
    int W_size = W_out/TILE_WIDTH, H_size = H_out/TILE_WIDTH;
    if (W_out%TILE_WIDTH)  W_size++;
    if (H_out%TILE_WIDTH)  H_size++;
    dim3 DimGrid(Map_out,W_size*H_size,Batch);
    dim3 DimBlock(TILE_WIDTH,TILE_WIDTH,1);
    // Set the kernel dimensions and call the kernel
    conv_forward_kernel<<<DimGrid,DimBlock>>>(device_output, device_input, device_mask, Batch, Map_out, Channel, Height, Width, K);
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Copy the output back to host
    int W_out = Width - K + 1, H_out = Height - K + 1;
    hipMemcpy(host_output, device_output, Batch * Map_out * W_out * H_out * sizeof(float), hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(device_input);
    hipFree(device_mask);
    hipFree(device_output);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
