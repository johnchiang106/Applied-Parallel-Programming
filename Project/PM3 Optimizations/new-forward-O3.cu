#include "hip/hip_runtime.h"
//O3 Sweeping various parameters to find best values (block sizes, amount of thread coarsening) (1 point)
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_WIDTH 8
#define MAX_NUM_THREADS 256
#define MASK_WIDTH 7
#define CHANNEL 4
#define MAP_SIZE 16
#define SM_IN (TILE_WIDTH + MASK_WIDTH - 1)
__constant__ float Const_Mask [MAP_SIZE*CHANNEL*MASK_WIDTH*MASK_WIDTH];

__global__ void conv_forward_kernel(float *output, const float *input, const float *mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    Batch - batch_size (number of images in x)
    Map_out - number of output feature maps
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */
    __shared__ float SM_Input [CHANNEL*SM_IN*SM_IN];

    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    int W_size = ceil(1.0 * Width_out / TILE_WIDTH);

    // Insert your GPU convolution kernel code here
    int tx = threadIdx.x, ty = threadIdx.y;
    int m = threadIdx.z;
    int h_offset = blockIdx.y * TILE_WIDTH;
    int w_offset = blockIdx.x * TILE_WIDTH;
    int b = blockIdx.z;
    //100, 4, 1, 86, 86, 7
    //100, 16, 4, 40, 40, 7

    const int InputSize = Batch * Channel * Height * Width;
    const int SM_Width = TILE_WIDTH + K - 1;
    const int msize = Channel * K * K;

    #define mask_4d(m, c, h, w) Const_Mask[(m) * (msize) + (c) * (K * K) + (h) * (K) + w]
    #define out_4d(b, m, h, w) output[(b) * (Map_out * Height_out * Width_out) + (m) * (Height_out * Width_out) + (h) * (Width_out) + w]
    #define input_idx(b, c, h, w) ((b) * (Channel * Height * Width) + (c) * (Height * Width) + (h) * (Width) + w)
    #define sm_in_idx(c, h, w) ((c)*(SM_Width*SM_Width) + (h)*(SM_Width) + w)

    // shared memory subinput
    // SM_Width = TILE_WIDTH + K - 1;
    // Restriction: TILE_WIDTH^2*Map_out > SM_Width^2
    int newIdx = m*TILE_WIDTH*TILE_WIDTH + ty*TILE_WIDTH + tx;
    int newY = newIdx / SM_Width;
    int newX = newIdx % SM_Width;
    if(newIdx < SM_Width*SM_Width){
        for (int c = 0; c < Channel; c++) {
            int sm_index = sm_in_idx(c,newY,newX);
            int index = input_idx(b,c,newY+h_offset,newX+w_offset);
            if(index < InputSize)
                SM_Input[sm_index] = input[index];
        }
    }
    __syncthreads();
    
    if(h_offset+ty < Height_out && w_offset+tx < Width_out){
        float acc = 0.0f;
        for (int c = 0; c < Channel; c++) { // sum over all input channels
            for (int p = 0; p < K; p++){ // loop over KxK filter
                for (int q = 0; q < K; q++){
                    acc += SM_Input[sm_in_idx(c, ty+p, tx+q)] * mask_4d(m, c, p, q);
                }
            }
        }
        out_4d(b, m, h_offset+ty, w_offset+tx) = acc;
    }

    #undef out_4d
    #undef mask_4d
    #undef input_idx
    #undef sm_in_idx
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    int W_out = Width - K + 1, H_out = Height - K + 1;
    // Allocate memory and copy over the relevant data structures to the GPU
    hipMalloc((void **) device_input_ptr, Batch * Channel * Height * Width * sizeof(float));
    hipMalloc((void **) device_mask_ptr, Channel * Map_out * K * K * sizeof(float));
    hipMalloc((void **) device_output_ptr, Batch * Map_out * W_out * H_out * sizeof(float));
    
    // We pass double pointers for you to initialize the relevant device pointers, which are passed to the other two functions.
    hipMemcpy(*device_input_ptr, host_input, Batch * Channel * Height * Width * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(Const_Mask), host_mask, Map_out * Channel * K * K * sizeof(float));
    // for(int i = 0; i < s; ++i){
    //     printf("%f, ", Const_Mask[i]);
    // }

    // }
    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }

}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    int W_out = Width - K + 1, H_out = Height - K + 1;
    int W_size = ceil(1.0 * W_out / TILE_WIDTH);
    int H_size = ceil(1.0 * H_out / TILE_WIDTH);

    dim3 DimGrid(W_size,H_size,Batch);
    // dim3 DimGrid(Map_out,W_size*H_size,Batch);
    dim3 DimBlock(TILE_WIDTH,TILE_WIDTH,Map_out);
    // Set the kernel dimensions and call the kernel
    conv_forward_kernel<<<DimGrid,DimBlock>>>(device_output, device_input, device_mask, Batch, Map_out, Channel, Height, Width, K);
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Copy the output back to host
    int W_out = Width - K + 1, H_out = Height - K + 1;
    hipMemcpy(host_output, device_output, Batch * Map_out * W_out * H_out * sizeof(float), hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(device_input);
    hipFree(device_mask);
    hipFree(device_output);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
