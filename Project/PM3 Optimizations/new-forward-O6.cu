#include "hip/hip_runtime.h"
//O6 Input channel reduction: tree (3 point)
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_WIDTH 16
#define MAX_NUM_THREADS 256
#define MASK_WIDTH 7
#define CHANNEL 4
#define MAP_SIZE 16
#define SM_IN (TILE_WIDTH + MASK_WIDTH - 1)
__constant__ float Const_Mask [MAP_SIZE*CHANNEL*MASK_WIDTH*MASK_WIDTH];

__global__ void conv_forward_kernel(float *output, const float *input, const float *mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    __shared__ float SM_Input [CHANNEL*SM_IN*SM_IN];
    __shared__ float SM_Output [CHANNEL][TILE_WIDTH][TILE_WIDTH];

    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    int W_size = ceil(1.0 * Width_out / TILE_WIDTH);

    // Insert your GPU convolution kernel code here
    int tx = threadIdx.x, ty = threadIdx.y, tz = threadIdx.z;
    int m = blockIdx.x;
    int h_offset = (blockIdx.y / W_size) * TILE_WIDTH;
    int w_offset = (blockIdx.y % W_size) * TILE_WIDTH;
    int b = blockIdx.z;
    //100, 4, 1, 86, 86, 7
    //100, 16, 4, 40, 40, 7

    const int BlockSize = TILE_WIDTH * TILE_WIDTH;
    const int InputSize = Batch * Channel * Height * Width;
    const int msize = Channel * K * K;
    const int SM_Width = TILE_WIDTH + K - 1;
    const int SM_ChSize = SM_Width * SM_Width;
    const int SM_InputSize = Channel * SM_ChSize;

    #define mask_4d(m, c, h, w) Const_Mask[(m) * (msize) + (c) * (K * K) + (h) * (K) + w]
    #define out_4d(b, m, h, w) output[(b) * (Map_out * Height_out * Width_out) + (m) * (Height_out * Width_out) + (h) * (Width_out) + w]
    #define input_idx(b, c, h, w) ((b) * (Channel * Height * Width) + (c) * (Height * Width) + (h) * (Width) + w)
    #define sm_in_idx(c, h, w) ((c)*(SM_ChSize) + (h)*(SM_Width) + w)

    SM_Output[tz][ty][tx] = 0;
    __syncthreads();

    // shared memory subinput
    for(int newIdx = ty*TILE_WIDTH + tx; newIdx < SM_ChSize; newIdx += BlockSize){
        int newY = newIdx / SM_Width;
        int newX = newIdx % SM_Width;
        int index = input_idx(b,tz,newY+h_offset,newX+w_offset);
        if(index < InputSize)
            SM_Input[sm_in_idx(tz,newY,newX)] = input[index];
    }
    __syncthreads();

    // if(m == 0 && b == 0 && h_offset == 0 && w_offset == 0){
    //     int s = Map_out * Channel * K * K;
        // for(int i = 0; i < s; ++i){
        //     printf("%f, ", mask[i]);
        // }
        // printf("\n");
    //     s = MAP_SIZE*CHANNEL*MASK_WIDTH*MASK_WIDTH;
    //     for(int i = 0; i < s; ++i){
    //         printf("%f, ", Const_Mask[i]);
    //     }
    //     printf("\n");
    // }
    // __syncthreads();
    int h = h_offset + ty, w = w_offset + tx;
    if(h < Height_out && w < Width_out){
        float acc = 0.0f;
        for (int p = 0; p < K; p++){ // loop over KxK filter
            for (int q = 0; q < K; q++){
                acc += SM_Input[sm_in_idx(tz, ty+p, tx+q)] * mask_4d(m, tz, p, q);
            }
        }
        // int offset = tz*TILE_WIDTH*TILE_WIDTH + ty*TILE_WIDTH + tx;
        // SM_Output[offset] = acc;
        SM_Output[tz][ty][tx] = acc;
        __syncthreads();

        // if(tz == 0){
        //     for(int i = 1; i < Channel; i++){
        //         SM_Output[0][ty][tx] += SM_Output[i][ty][tx];
        //     }
        // }

        //Restriction: Channel is power of 2
        for (unsigned int stride = Channel / 2; stride >= 1; stride /= 2) {
            __syncthreads();
            if (tz < stride && tz + stride < Channel){
                SM_Output[tz][ty][tx] += SM_Output[tz+stride][ty][tx];
                // SM_Output[offset] += SM_Output[offset + stride * TILE_WIDTH * TILE_WIDTH];
            }
        }
        __syncthreads();
        if(tz == 0){
            out_4d(b, m, h, w) = SM_Output[0][ty][tx];
        }
    }

    #undef out_4d
    #undef mask_4d
    #undef input_idx
    #undef sm_in_idx
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    int W_out = Width - K + 1, H_out = Height - K + 1;
    // Allocate memory and copy over the relevant data structures to the GPU
    hipMalloc((void **) device_input_ptr, Batch * Channel * Height * Width * sizeof(float));
    hipMalloc((void **) device_mask_ptr, Channel * Map_out * K * K * sizeof(float));
    hipMalloc((void **) device_output_ptr, Batch * Map_out * W_out * H_out * sizeof(float));
    
    // We pass double pointers for you to initialize the relevant device pointers, which are passed to the other two functions.
    hipMemcpy(*device_input_ptr, host_input, Batch * Channel * Height * Width * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(Const_Mask), host_mask, Map_out * Channel * K * K * sizeof(float));

    // }
    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }

}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    int W_out = Width - K + 1, H_out = Height - K + 1;
    int W_size = ceil(1.0 * W_out / TILE_WIDTH);
    int H_size = ceil(1.0 * H_out / TILE_WIDTH);

    dim3 DimGrid(Map_out,W_size*H_size,Batch);
    // dim3 DimGrid(1,W_size*H_size,Batch);
    dim3 DimBlock(TILE_WIDTH,TILE_WIDTH,Channel);
    // Set the kernel dimensions and call the kernel
    conv_forward_kernel<<<DimGrid,DimBlock>>>(device_output, device_input, device_mask, Batch, Map_out, Channel, Height, Width, K);
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Copy the output back to host
    int W_out = Width - K + 1, H_out = Height - K + 1;
    hipMemcpy(host_output, device_output, Batch * Map_out * W_out * H_out * sizeof(float), hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(device_input);
    hipFree(device_mask);
    hipFree(device_output);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
